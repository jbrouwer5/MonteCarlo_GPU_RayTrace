#include<stdio.h>
#include <iostream>
#include <fstream>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<hiprand/hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include <sstream>
#include <chrono>
#include <mutex>

// helper function to print the array to a file
void print_matrix(int size, double* matrix) {
    std::stringstream ss; // Use stringstream to buffer output
    for (int i = 0; i < size*size; i++) {
        ss << matrix[i];
        if (i < size*size - 1) 
            ss << ",";
    }
    ss << "\n";

    // Now write to file in one go
    std::ofstream myFile("output.txt", std::ios::app); // Open file in append mode
    myFile << ss.str();
    myFile.close(); 
}

__global__ void trace(double *, hiprandState *, hiprandState *, unsigned long long int*, int); /* device function */

__global__ void trace(double *G, hiprandState *phiStates, hiprandState *thetaStates, unsigned long long int *count, int N){
     
    int c[] = {0,12,0}; int l[] = {4,4,-1}; // x,y,z
    int radius = 6; double w[] = {0,2,0}; int wMax = 2; 
    double cellSize = wMax*2.0 / double(N);
    double v[3]; double phi; double cosTheta; double sinTheta; 
    double t; double i[3];  
    double b; int row; int col; 
    double inter; int cSquare = c[0]*c[0] + c[1]*c[1] + c[2]*c[2]; 
    double magLSubI; double magISubC; 
    double n[3]; double s[3]; 

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int intCount = 0; intCount<500; intCount++){

        do {
            // atomicAdd(count,1);
            phi = hiprand_uniform_double(&phiStates[tid]) * M_PI;
            cosTheta = hiprand_uniform_double(&thetaStates[tid]) * 2.0 - 1.0; 
            sinTheta = sqrt(1 - pow(cosTheta,2));

            v[0] = sinTheta * cos(phi); 
            v[1] = sinTheta * sin(phi); 
            v[2] = cosTheta; 

            w[0] = (w[1] / v[1]) * v[0]; 
            w[2] = (w[1] / v[1]) * v[2]; 

            inter = v[0]*c[0] + v[1]*c[1] + v[2]*c[2];
            t = inter*inter + radius*radius - cSquare;
        }
        while (!(abs(w[0]) < wMax && abs(w[2]) < wMax && t > 0));
        
        t = (v[0]*c[0] + v[1]*c[1] + v[2]*c[2]) - sqrt(t); 
        
        // i = t*v
        i[0] = t * v[0]; i[1] = t * v[1]; i[2] = t * v[2]; 

        // |i-c|
        magISubC = sqrt((i[0]-c[0])*(i[0]-c[0]) + (i[1]-c[1])*(i[1]-c[1]) + (i[2]-c[2])*(i[2]-c[2])); 

        // // n = (i-c) / |i-c|
        n[0] = (i[0]-c[0]) / magISubC; 
        n[1] = (i[1]-c[1]) / magISubC; 
        n[2] = (i[2]-c[2]) / magISubC; 

        // |l-i|
        magLSubI = sqrt((l[0]-i[0])*(l[0]-i[0]) + (l[1]-i[1])*(l[1]-i[1]) + (l[2]-i[2])*(l[2]-i[2])); 

        // // s = (l-i) / |l-i|
        s[0] = (l[0]-i[0]) / magLSubI;
        s[1] = (l[1]-i[1]) / magLSubI;
        s[2] = (l[2]-i[2]) / magLSubI;

        // b = max(0, s*n)
        b = (s[0]*n[0] + s[1]*n[1] + s[2]*n[2]);

        if (b > 0){
            // find (i, j) such that G(i, j) is the gridpoint of⃗ W on G 
            // use wX and wZ to calculate the point on the grid 
            row = floor((w[0]+wMax) / cellSize); 
            col = floor((w[2]+wMax) / cellSize); 
            
            atomicAdd(&G[row*N+col], b);
        }
    }
}

__global__ void initCurandStates(hiprandState *phiStates, hiprandState *thetaStates) {
    int seed1 = 0; 
    int seed2 = 1; 
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed1, tid, 0, &phiStates[tid]);
    hiprand_init(seed2, tid, 0, &thetaStates[tid]);
}

int main(int argc, char **argv){

    if (argc < 5)
    {
        std::cout << "Not enough arguments"; 
        return 0; 
    }
    
    // usage : raytrace nrays ngrid nblocks ntpb
    int numRays = std::stoi(argv[1]);
    int N = std::stoi(argv[2]); 
    int nBlocks = std::stoi(argv[3]);
    int ntpb = std::stoi(argv[4]); 

    auto start = std::chrono::steady_clock::now();

    int i;
    double *G = new double[N*N]; 
    double *dev_G; 
    unsigned long long int*count = new unsigned long long int; 
    *count = 0; 
    unsigned long long int* dev_count;

    hiprandState *d_phiStates, *d_thetaStates;

    for (i = 0; i < N*N; ++i){
        G[i] = 0; 
    }
    
    hipMalloc( (void **) &dev_G, N*N*sizeof(double));
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        std::cout << "CUDA error1: " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }
    hipMemcpy(dev_G, G, N*N*sizeof(double), hipMemcpyHostToDevice);
    error = hipGetLastError();
    if(error != hipSuccess) {
        std::cout << "CUDA error2: " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }

    hipMalloc( (void **) &dev_count, sizeof(unsigned long long int));
    error = hipGetLastError();
    if(error != hipSuccess) {
        std::cout << "CUDA error3: " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }
    hipMemcpy(dev_count, count, sizeof(unsigned long long int), hipMemcpyHostToDevice);
    error = hipGetLastError();
    if(error != hipSuccess) {
        std::cout << "CUDA error4: " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }
    
    auto kstart = std::chrono::steady_clock::now();
    
    // Allocate memory for curandStates on device
    hipMalloc(&d_phiStates, nBlocks*ntpb * sizeof(hiprandState));
    hipMalloc(&d_thetaStates, nBlocks*ntpb * sizeof(hiprandState));

    // Initialize curandStates
    initCurandStates<<<nBlocks, ntpb>>>(d_phiStates, d_thetaStates);
    
    hipDeviceSynchronize();
    trace<<<nBlocks, ntpb>>>(dev_G, d_phiStates, d_thetaStates, dev_count, N);
    error = hipGetLastError();
    if(error != hipSuccess) {
        std::cout << "CUDA error5: " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }

    hipDeviceSynchronize();
    std::cout << "Kernel took " << std::chrono::duration_cast<std::chrono::milliseconds>(
            std::chrono::steady_clock::now() - kstart).count() << " seconds" << std::endl;
    
    hipMemcpy(G, dev_G, N*N*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(count, dev_count, sizeof(long), hipMemcpyDeviceToHost);
    std::cout << "total rays is " << *count << std::endl; 
    print_matrix(N, G);

    delete[] G;
    hipFree(dev_G); 
    hipFree(d_phiStates);
    hipFree(d_thetaStates);
    delete count; 
    hipFree(dev_count); 

    std::cout << "Took " << std::chrono::duration_cast<std::chrono::milliseconds>(
            std::chrono::steady_clock::now() - start).count() << " seconds" << std::endl;

    exit(0);
}
